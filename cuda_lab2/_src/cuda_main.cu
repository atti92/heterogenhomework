#include "hip/hip_runtime.h"
#include <stdio.h>
#include "time.h"
#include "omp.h"

#include "emmintrin.h"
#include "nmmintrin.h"

#ifdef __INTELLISENSE__
	#ifndef __HIPCC__ 
		#define __HIPCC__
	#endif
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "math_functions.h"

#define N 512
#define RUNS 200

#define USE_STREAM 0

__constant__ int filter_laplace[5][5] = { 2, 4, 5, 4, 2,
4, 9, 12, 9, 4,
5, 12, 15, 12, 5,
4, 9, 12, 9, 4,
2, 4, 5, 4, 2 };

__constant__ float filter_laplace_f[5][5] = { 2.0f / 159, 4.0f / 159, 5.0f / 159, 4.0f / 159, 2.0f / 159,
4.0f / 159, 9.0f / 159, 12.0f / 159, 9.0f / 159, 4.0f / 159,
5.0f / 159, 12.0f / 159, 15.0f / 159, 12.0f / 159, 5.0f / 159,
4.0f / 159, 9.0f / 159, 12.0f / 159, 9.0f / 159, 4.0f / 159,
2.0f / 159, 4.0f / 159, 5.0f / 159, 4.0f / 159, 2.0f / 159 };



/*
 * most is kibővített kép van, 2-2 pixellel szélesebb és magasabb
 * egymástól független dolgokat kellene számolniuk, kimeneti pixelek eléggé függetlenek
 * threadIdx a blokkban, blockIdx a blokk helye -> innen a row és col (kibővített képen eltolással)
 */
__global__ void kernel_conv(unsigned char* gInput, unsigned char* gOutput, int imgWidthF)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y + 2;
	int col = blockIdx.x * blockDim.x + threadIdx.x + 2;
	int pixel[3] = { 0, 0, 0 };

	// írási cím: (row*imgWidthF+col)*3
	for (int i = 0; i < 5; i++) {
		for (int j = 0; j < 5; j++) {
			for (int rgb = 0; rgb < 3; rgb++) {
				pixel[rgb] += gInput[((row + i - 2)*imgWidthF + (col + j - 2)) * 3 + rgb] * filter_laplace[i][j] / 159;
			}
		}
	}
	for (int rgb = 0; rgb < 3; rgb++) {
		pixel[rgb] = max(0, min(pixel[rgb], 255));
		pixel[rgb] = gInput[(row*imgWidthF + col) * 3 + rgb] - pixel[rgb];
		pixel[rgb] = max(0, min(pixel[rgb], 255));
		pixel[rgb] = gInput[(row*imgWidthF + col) * 3 + rgb] + pixel[rgb];
		pixel[rgb] = max(0, min(pixel[rgb], 255));
		gOutput[(row*imgWidthF + col) * 3 + rgb] = pixel[rgb];
	}
}

__global__ void kernel_conv_shrmem(unsigned char* gInput, unsigned char* gOutput, int imgWidthF)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y + 2;
	int col = blockIdx.x * blockDim.x + threadIdx.x + 2;

	// base cím a cache töltéséhez
	int blockRow = blockIdx.y * blockDim.y;
	int blockCol = blockIdx.x * blockDim.x;

	// 20x20 méretű blokk és 3 csatorna
	__shared__ unsigned char blockData[20][20][3];

	// 1 dimeneziós 
	int id1d = threadIdx.y * 16 + threadIdx.x;
	if (id1d < 60) {
		// első 60 szál tölti az első sort, aztán a másodikat stb. összesen 60 bájt egy sor
		for (int cacheRow = 0; cacheRow < 20; cacheRow++) {
			// blockRow*imgWidthF+blockCol a 20x20-as cache blokk bal felső sarka az eredeti képen
			// cache soronként kell ebből lefelé menni (imgWidthF szorzás); és a 3 csatorna miatt x3
			// a sorokban a 60 bájtot már tudja kezelni az id1d változó
			blockData[cacheRow][id1d / 3][id1d % 3] = gInput[((blockRow + cacheRow)*imgWidthF + blockCol) * 3 + id1d];
		}
	}

	// mindenképp várni kell, mert a többi szál továbbmenne, ami nem tölt éppen
	__syncthreads();

	int pixel[3] = { 0, 0, 0 };
	for (int i = 0; i < 5; i++) {
		for (int j = 0; j < 5; j++) {
			for (int rgb = 0; rgb < 3; rgb++) {
				// threadIdx 16x16-ban mozog, a blockData 20x20 -> így 5x5 kernellel pont jól címzi
				pixel[rgb] += blockData[threadIdx.y + i][threadIdx.x + j][rgb] * filter_laplace[i][j] / 159;
			}
		}
	}
	for (int rgb = 0; rgb < 3; rgb++) {
		pixel[rgb] = max(0, min(pixel[rgb], 255));
		pixel[rgb] = blockData[threadIdx.y + 2][threadIdx.x + 2][rgb] - pixel[rgb];
		pixel[rgb] = max(0, min(pixel[rgb], 255));
		pixel[rgb] = blockData[threadIdx.y + 2][threadIdx.x + 2][rgb] + pixel[rgb];
		pixel[rgb] = max(0, min(pixel[rgb], 255));
		gOutput[(row*imgWidthF + col) * 3 + rgb] = pixel[rgb];
	}
}

// float accu, uint shmem
__global__ void kernel_convf_shrmem(unsigned char* gInput, unsigned char* gOutput, unsigned int imgWidthF)
{
	unsigned int row = blockIdx.y * blockDim.y + threadIdx.y + 2;
	unsigned int col = blockIdx.x * blockDim.x + threadIdx.x + 2;

	// base cím a cache töltéséhez
	unsigned int blockRow = blockIdx.y * blockDim.y;
	unsigned int blockCol = blockIdx.x * blockDim.x;

	// 20x20 méretű blokk és 3 csatorna
	__shared__ unsigned char blockData[3][20][20];

	// 1 dimeneziós 
	unsigned int id1d = threadIdx.y * 16 + threadIdx.x;
	if (id1d < 60) {
		// első 60 szál tölti az első sort, aztán a másodikat stb. összesen 60 bájt egy sor
		for (unsigned int cacheRow = 0; cacheRow < 20; cacheRow++) {
			// blockRow*imgWidthF+blockCol a 20x20-as cache blokk bal felső sarka az eredeti képen
			// cache soronként kell ebből lefelé menni (imgWidthF szorzás); és a 3 csatorna miatt x3
			// a sorokban a 60 bájtot már tudja kezelni az id1d változó
			blockData[id1d % 3][cacheRow][id1d / 3] = gInput[((blockRow + cacheRow)*imgWidthF + blockCol) * 3 + id1d];
		}
	}

	// mindenképp várni kell, mert a többi szál továbbmenne, ami nem tölt éppen
	__syncthreads();

	float pixel = 0;
	for (unsigned int rgb = 0; rgb < 3; rgb++)
	{
		pixel = 0;
		for (unsigned int i = 0; i < 5; i++) {
			for (unsigned int j = 0; j < 5; j++) {
				// threadIdx 16x16-ban mozog, a blockData 20x20 -> így 5x5 kernellel pont jól címzi
				pixel += blockData[rgb][threadIdx.y + i][threadIdx.x + j] * filter_laplace_f[i][j];
			}
		}
		gOutput[(row*imgWidthF + col) * 3 + rgb] = min(blockData[rgb][threadIdx.y + 2][threadIdx.x + 2] + max(0.f, blockData[rgb][threadIdx.y + 2][threadIdx.x + 2] - max(0.f, min(pixel, 255.f))), 255.f);
	}
}

// float accu, float shmem
__global__ void kernel_convf_shrmemf(unsigned char* gInput, unsigned char* gOutput, int imgWidthF)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y + 2;
	int col = blockIdx.x * blockDim.x + threadIdx.x + 2;

	// base cím a cache töltéséhez
	int blockRow = blockIdx.y * blockDim.y;
	int blockCol = blockIdx.x * blockDim.x;

	// 20x20 méretű blokk és 3 csatorna
	__shared__ float blockData[3][20][20];

	// 1 dimeneziós 
	unsigned int id1d = threadIdx.y * 16 + threadIdx.x;
	if (id1d < 60) {
		// első 60 szál tölti az első sort, aztán a másodikat stb. összesen 60 bájt egy sor
		for (int cacheRow = 0; cacheRow < 20; cacheRow++) {
			// blockRow*imgWidthF+blockCol a 20x20-as cache blokk bal felső sarka az eredeti képen
			// cache soronként kell ebből lefelé menni (imgWidthF szorzás); és a 3 csatorna miatt x3
			// a sorokban a 60 bájtot már tudja kezelni az id1d változó
			
			blockData[id1d % 3][cacheRow][id1d/3] = gInput[((blockRow + cacheRow)*imgWidthF + blockCol) * 3 + id1d];
		}
	}

	// mindenképp várni kell, mert a többi szál továbbmenne, ami nem tölt éppen
	__syncthreads();

	float pixel[3] = { 0, 0, 0 };
	for (int i = 0; i < 5; i++) {
		for (int j = 0; j < 5; j++) {
			for (int rgb = 0; rgb < 3; rgb++) {
				// threadIdx 16x16-ban mozog, a blockData 20x20 -> így 5x5 kernellel pont jól címzi
				pixel[rgb] += blockData[rgb][threadIdx.y + i][threadIdx.x + j] * filter_laplace_f[i][j];
			}
		}
	}
	for (int rgb = 0; rgb < 3; rgb++) {
		pixel[rgb] = max(0.f, min(pixel[rgb], 255.f));
		pixel[rgb] = blockData[rgb][threadIdx.y + 2][threadIdx.x + 2] - pixel[rgb];
		pixel[rgb] = max(0.f, min(pixel[rgb], 255.f));
		pixel[rgb] = blockData[rgb][threadIdx.y + 2][threadIdx.x + 2] + pixel[rgb];
		pixel[rgb] = max(0.f, min(pixel[rgb], 255.f));
		gOutput[(row*imgWidthF + col) * 3 + rgb] = pixel[rgb];
	}
}

/*
 * hoszton futó kód, vannak benne cuda függvények is.
 * |R|G|B|R|G|B|R|G|B|, 8 bit unsigned számok sorban
 * GPU-n foglalt memória a kiterjesztett kép mérete
 */
void cudaMain(unsigned char *imgSrc, unsigned char *imgRes,
	int imgWidth, int imgHeight, int imgFOfssetW, int imgFOfssetH)
{
	clock_t s0, e0;
	double d0;

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	int imgWidthF = imgWidth + 2 * imgFOfssetW;

	// kiterjesztett kép mérete, +3 x komponenens mérete
	int size = (imgWidth + 2 * imgFOfssetW)*(imgHeight + 2 * imgFOfssetH)*sizeof(unsigned char) * 3;

	unsigned char *gInput, *gOutput;
	float *gCoeff;
	// GPU mem foglalás; bemenetre és kimenetre is, memória pointereket ad vissza
	hipMalloc((void**)&gInput, size);
	hipMalloc((void**)&gOutput, size);
	hipMalloc((void**)&gCoeff, 25 * sizeof(float));

	// kimeneti pixelek száma alapján; ezek lesznek párhuzamosítva
	// thread block-okból építünk gridet (blockban szálak)
	// összesen 1920x1200 pixel -> ennyi szál
	// blokkban elérhető a közös shared memória, őket lehet szinkronizálni expliciten
	// ha nem többszörösre jön ki a méret, akkor elágazás kell, és a szélső warpok lassabbak lesznek, a többi gyors
	dim3 thrBlock(16, 16);	// gpu függően 1024 stb. lehet. legkisebb csoport a warp, 32 szál, ezek SIMD módon mennek -> 32 többszöröse kell
	// első warp a 16x16-os blokk felső két sora
	// minden pixelt 25x olvasunk ki, cache segíti
	// beolvasható shared mem-be is, 16x16-os blokk 20x20 pixelt cache-elhet; 32 szál 32 különböző bankot olvasson (32 bank, 1 bank 4 bájt széles; ugyanazt a szót érdemes olvasni; első sor első bank lefed 0..14; 60 bájt)
	// kvázi folytonos címtartomány; 4 byte széles, b0|b1|...|b31|b0|...; unsigned char |0|1|2|3| @b0; 32x4 byte az első 32 bankban, 128. byte megint @bank0
	// 32 bank párhuzamosan olvasható, nagy sávszélesség, bank másik szavát ne bazgerálja másik szál. double v int64 esetén van bankütközés, itt nincs
	// ilyen tömb [y][x][rgb] -> [20][20][3]
	// 1. megoldás: sor=60 byte, első 60 szál betöltheti az első sort, 20 soron keresztül
	// 2.: 1. 60 szál és 2. 60 szál... összesen 4 sor egyszerre, 5-ös for ciklus kell csak

	// id1d = threadIdx.y*16+threadIdx.x a szálazonosító a blokkon belül, 60 alatt töltögethet; soronként teljes képszélességgel nő
	dim3 thrGrid(imgWidth / 16, imgHeight / 16);

	s0 = clock();
	hipMemcpy(gInput, imgSrc, size, hipMemcpyHostToDevice);
	for (int i = 0; i < RUNS; i++)
	{
		kernel_conv << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidthF);
		hipDeviceSynchronize();
	}
	e0 = clock();
	d0 = static_cast<double>(e0 - s0) / CLOCKS_PER_SEC;
	printf("total CPU TIME: %4.4fs\n", d0);
	printf("1 cycle CPU TIME: %4.4fms\n", d0 * 1000 / RUNS);

	s0 = clock();
	hipMemcpy(gInput, imgSrc, size, hipMemcpyHostToDevice);
	for (int i = 0; i < RUNS; i++)
	{
		kernel_conv_shrmem << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidthF);
		hipDeviceSynchronize();
	}
	e0 = clock();
	d0 = static_cast<double>(e0 - s0) / CLOCKS_PER_SEC;
	printf("total GPU TIME: %4.4fs\n", d0);
	printf("1 cycle GPU TIME: %4.4fms\n", d0 * 1000 / RUNS);

	
	s0 = clock();
	hipMemcpy(gInput, imgSrc, size, hipMemcpyHostToDevice);
	for (int i = 0; i < RUNS; i++)
	{
		kernel_convf_shrmem << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidthF);
		hipDeviceSynchronize();
	}
	e0 = clock();
	d0 = static_cast<double>(e0 - s0) / CLOCKS_PER_SEC;
	printf("total GPU TIME: %4.4fs\n", d0);
	printf("1 cycle GPU TIME: %4.4fms\n", d0 * 1000 / RUNS);


	s0 = clock();
	hipMemcpy(gInput, imgSrc, size, hipMemcpyHostToDevice);
	for (int i = 0; i < RUNS; i++)
	{
		kernel_convf_shrmemf << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidthF);
		hipDeviceSynchronize();
	}
	
	e0 = clock();
	d0 = static_cast<double>(e0 - s0) / CLOCKS_PER_SEC;
	printf("total GPU TIME: %4.4fs\n", d0);
	printf("1 cycle GPU TIME: %4.4fms\n", d0 * 1000 / RUNS);

	hipMemcpy(imgRes, gOutput, size, hipMemcpyDeviceToHost);
	hipFree(gInput); hipFree(gOutput);
	

	hipDeviceReset();
}