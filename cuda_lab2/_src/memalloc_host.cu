#include "memory.h"

#ifdef __INTELLISENSE__
#ifndef __CUDACC__ 
#define __CUDACC__
#endif
#endif

#include "hip/hip_runtime.h"


#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "math_functions.h"

#define USE_CUDA_MEMALLOC 0
#define USE_WC 0

void memalloc_host(int imgWidthF, int imgHeightF, unsigned char **input, unsigned char **output)
{
	hipError_t cuda_ret;

#if USE_CUDA_MEMALLOC == 0
	*input   = (unsigned char *)malloc(3*imgWidthF*imgHeightF*sizeof(unsigned char));
	*output = (unsigned char *)malloc(3*imgWidthF*imgHeightF*sizeof(unsigned char));
#else
	#if USE_WC == 0
		cuda_ret = cudaHostAlloc(input, 3*imgWidthF*imgHeightF*sizeof(unsigned char), cudaHostAllocDefault);
	#else
		cuda_ret = cudaHostAlloc(input, 3*imgWidthF*imgHeightF*sizeof(unsigned char), cudaHostAllocWriteCombined);
	#endif
	cuda_ret = cudaHostAlloc(output, 3*imgWidthF*imgHeightF*sizeof(unsigned char), cudaHostAllocDefault);

#endif
}

void memfree_host(unsigned char **input, unsigned char **output)
{
	hipError_t cuda_ret;

#if USE_CUDA_MEMALLOC == 0
	free(*input);
	free(*output);
#else
	cuda_ret = cudaFree(input);
	cuda_ret = cudaFree(output);
#endif
}